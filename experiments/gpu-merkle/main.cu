#include "sha256.h"
#include <map>
#include <hip/hip_runtime.h>
#include <stdio.h>

std::map<int, uint32_t *> context_idata[2];
std::map<int, uint32_t *> context_odata[2];
std::map<int, hipStream_t> context_streams[2];
std::map<int, uint32_t *> context_tstate[2];
std::map<int, uint32_t *> context_ostate[2];
std::map<int, uint32_t *> context_hash[2];

#define HASH_SIZE (8 * sizeof(uint32_t))

int main(int argc, const char* argv[]) {
    int thrd_id = 0;
    int throughput = 128;
    int stream = 0;
    uint32_t cpu_pdata[20] = {0};
    uint32_t cpu_midstate[8] = {0};

    cpu_pdata[0] = 2;

    context_idata[stream][0] = NULL;
    hipMalloc(&context_idata[0][0], 32 * sizeof(uint32_t));
    hipMemset(&context_idata[0][0], strtol(argv[1], nullptr, 10), 32 * sizeof(uint32_t));

    context_odata[stream][0] = NULL;
    hipMalloc(&context_odata[0][0], 32 * sizeof(uint32_t));

    context_ostate[stream][0] = NULL;
    hipMalloc(&context_ostate[stream][0], 32 * sizeof(uint32_t));

    context_tstate[stream][0] = NULL;
    hipMalloc(&context_tstate[0][0], 32 * sizeof(uint32_t));
 
    context_hash[stream][0] = NULL;
    hipMalloc(&context_hash[stream][0], 8 * sizeof(uint32_t));

    hipStream_t cudaStream;
    hipStreamCreate(&cudaStream);
    context_streams[stream][0] = cudaStream;

    printf("prepare_sha256:\n");
    prepare_sha256(thrd_id, cpu_pdata, cpu_midstate);
    printf("pre_sha256\n");
    pre_sha256(thrd_id, stream, 0, throughput);
    printf("post_sha256\n");
    post_sha256(thrd_id, stream, throughput);

    uint32_t* h_hash = (uint32_t*)malloc(HASH_SIZE);

    hipMemcpy(h_hash, context_hash[stream][thrd_id], HASH_SIZE, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    for (int i = 0; i < HASH_SIZE / sizeof(uint32_t); i++) {
        printf("%08x ", h_hash[i]);
    }
    printf("\n");
 
    return 0;
}
