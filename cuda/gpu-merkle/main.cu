#include "sha256.h"
#include <map>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "cuda_util.h"

std::map<int, uint32_t *> context_idata[2];
std::map<int, uint32_t *> context_odata[2];
std::map<int, hipStream_t> context_streams[2];
std::map<int, uint32_t *> context_tstate[2];
std::map<int, uint32_t *> context_ostate[2];
std::map<int, uint32_t *> context_hash[2];

#define HASH_SIZE (8 * sizeof(uint32_t))

int main(int argc, const char* argv[]) {
    int thrd_id = 0;
    int throughput = 128;
    int stream = 0;
    //uint32_t h_pdata[20] = {0};
    int num_blocks = 128;
    uint32_t* h_pdata = (uint32_t*)calloc(num_blocks * 16, sizeof(uint32_t));
    uint32_t cpu_midstate[8] = {0};

    if (argc != 3) {
        printf("Usage: gpumerkle <init value> <num bytes>\n");
        return 1;
    }

    fprintf(stderr, "starting:\n");
    int input = strtol(argv[1], nullptr, 10);
    int input2 = strtol(argv[2], nullptr, 10);
    h_pdata[1] = input;

    context_idata[stream][0] = NULL;
    hipMalloc(&context_idata[stream][0], 32 * sizeof(uint32_t));
    //hipMemset(&context_idata[stream][0], strtol(argv[1], nullptr, 10), 32 * sizeof(uint32_t));

    context_odata[stream][0] = NULL;
    hipMalloc(&context_odata[stream][0], 32 * sizeof(uint32_t));
    //hipMemset(&context_odata[stream][0], strtol(argv[1], nullptr, 10), 32 * sizeof(uint32_t));

    context_ostate[stream][0] = NULL;
    hipMalloc(&context_ostate[stream][0], 32 * sizeof(uint32_t));

    context_tstate[stream][0] = NULL;
    hipMalloc(&context_tstate[0][0], 32 * sizeof(uint32_t));
 
    context_hash[stream][0] = NULL;
    hipMalloc(&context_hash[stream][0], 8 * sizeof(uint32_t));

    uint32_t* d_hash = NULL;
    hipMalloc(&d_hash, num_blocks * 8 * sizeof(uint32_t));

    hipStream_t cudaStream;
    hipStreamCreate(&cudaStream);
    context_streams[stream][0] = cudaStream;

    memset(h_pdata, input, input2 * sizeof(uint32_t));
    uint32_t* d_pdata = nullptr;
    hipMalloc(&d_pdata, 20 * sizeof(uint32_t));
    checkCudaErrors(hipMemcpy(d_pdata, h_pdata, 20 * sizeof(uint32_t), hipMemcpyHostToDevice));
    //hipMemset(d_pdata, strtol(argv[1], nullptr, 10), 20 * sizeof(uint32_t));
    memset(h_pdata, 0, 20 * sizeof(uint32_t));
    hipMemcpy(h_pdata, d_pdata, 20 * sizeof(uint32_t), hipMemcpyDeviceToHost);
    for (int i = 0; i < 20; i++) {
        printf("%x ", h_pdata[i]);
    }
    printf("\n");

    int num_levels = 2;
    int levels[] = {2, 2};
    sha256_merkle(d_pdata, d_hash, levels, num_levels);

    uint32_t* h_hash = (uint32_t*)calloc(num_blocks * HASH_SIZE, 1);

    hipMemcpy(h_hash, context_hash[stream][thrd_id], HASH_SIZE, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    for (int i = 0; i < (num_blocks * HASH_SIZE) / sizeof(uint32_t); i++) {
        printf("%08x ", h_hash[i]);
    }
    printf("\n");
 
    return 0;
}
